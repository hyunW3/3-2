#include <hip/hip_runtime.h>

__host__ void counting_sort(int arr[], int size, int max_val)
{
   // fill in 
   int * arr_d,histogram_d;
   int i,j;
   int index =0;
   hipMalloc((void**)&arr_d,size);
   hipMemcpy(arr_d,arr,size,hipMemcpyHostToDevice);
   hipMalloc((void**)&histogram_d,max_val);
   //for(i=0; i<max_val; i++){
      histogram_d[i] =0;
   //}
   //for (i=0; i<size; i++){
      histogram_d[arr_d[i]]++;
   //}

   //for(i=0; i<max_val; i++){
   //   for(j=0; j<histogram_d[i]; j++){
         arr_d[index++] = i;
   //   }
   //}

   // copy to host & finsh
   hipMemcpy(arr,arr_d,size,hipMemcpyDeviceToHost);
   hipFree(arr_d); hipFree(histogram_d);
}

