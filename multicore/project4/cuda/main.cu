#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<10,10>>>();
    hipDeviceSynchronize();
    return 0;
}

