#include <hip/hip_runtime.h>
#include <cmath>
   
int nBlk,nTid;
__global__ void cuda_sort(int* arr_d,int* out, int* histogram_d, int size, int max_val);
__host__ void counting_sort(int arr[], int size, int max_val)
{
   int* arr_d;
   int* out;
   int* histogram_d;
   hipMalloc((void**)&arr_d,size*sizeof(int));
   hipMalloc((void**)&out  ,size*sizeof(int));
   hipMemcpy(arr_d,arr,size*sizeof(int),hipMemcpyHostToDevice);
   hipMalloc((void**)&histogram_d,max_val*sizeof(int));
   hipMemset((void**)&histogram_d,0,max_val*sizeof(int));
   if(size>1024)  cuda_sort<<<ceil(size/1024),1024,size*sizeof(int)>>>(arr_d,out, histogram_d, size, max_val);
   else           cuda_sort<<<1,size,size*sizeof(int)>>>(arr_d,out, histogram_d, size, max_val);
   // copy to host & finsh
   hipMemcpy(arr,out,size*sizeof(int),hipMemcpyDeviceToHost);
   hipFree(arr_d); hipFree(histogram_d);
   hipFree(out);
}

__global__ void cuda_sort(int* arr_d,int* out, int* histogram_d, int size, int max_val){
   extern __shared__ int position[]; 
   int i = threadIdx.x + blockDim.x * blockIdx.x;
   //__device__ int histogram_d[max_val];
   if(i<size){
      atomicAdd(&histogram_d[arr_d[i]],1);
   }
   __syncthreads();
   if(i<max_val){
      if(i==0) position[0] = 0;
      else {
         position[i] = histogram_d[i-1];
         for(int j=0; j<i-1; j++){
            position[i] += histogram_d[j];
         }
      }
   }
   __syncthreads();
   if(i<max_val){ 

   for(int j=0; j<histogram_d[i]; j++){
      out[position[i]+j] = i;
   }

   //   memset(&arr_d[position[i]],i,sizeof(int)*histogram_d[i]);
   }
   // device code
}


