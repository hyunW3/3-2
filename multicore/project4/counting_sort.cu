#include <hip/hip_runtime.h>
int nBlk,nTid;
__global__ void cuda_sort(int* arr_d,int* histogram_d, int size, int max_val);
__host__ void counting_sort(int arr[], int size, int max_val)
{
   nTid = 512;
   nBlk = size/nTid;
   if(nBlk * nTid < size){
      nBlk +=1;
   }
   int * arr_d;
   int* histogram_d;
   hipMalloc((void**)&arr_d,size*sizeof(int));
   hipMemcpy(arr_d,arr,size*sizeof(int),hipMemcpyHostToDevice);
   hipMalloc((void**)&histogram_d,max_val*sizeof(int));
   hipMemset((void**)&histogram_d,0,max_val*sizeof(int));
   //cuda_sort<<<nBlk,nTid>>>(arr_d,histogram_d, size, max_val);
   cuda_sort<<<1,size,size*sizeof(int)>>>(arr_d,histogram_d, size, max_val);
   // copy to host & finsh
   hipMemcpy(arr,arr_d,size*sizeof(int),hipMemcpyDeviceToHost);
   hipFree(arr_d); hipFree(histogram_d);
}
__global__ void cuda_sort(int* arr_d,int* histogram_d, int size, int max_val){
   extern __shared__ int position[]; 
   int i = threadIdx.x + blockDim.x * blockIdx.x;
   //__device__ int histogram_d[max_val];
   if(i<size){
      atomicAdd(&histogram_d[arr_d[i]],1);
   }
   __syncthreads();
   if(i<max_val){
      position[i] = histogram_d[i-1];
      for(int j=0; j<i-1; j++){
         position[i] += histogram_d[j];
      }
   }
   __syncthreads();
   if(i<max_val){
   //   arr_d[i] = histogram_d[i];
   //   arr_d[i] = position[i];
   for(int j=0; j<histogram_d[i]; j++){
      arr_d[position[i]+j] = i;
   }
   
   }
   // device code
}


