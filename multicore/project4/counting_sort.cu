#include <hip/hip_runtime.h>
int nBlk,nTid;
__device__ int position[10]={0,};
__global__ void cuda_sort(int* arr_d,int* histogram_d, int size, int max_val);
__host__ void counting_sort(int arr[], int size, int max_val)
{
   nTid = 512;
   nBlk = size/nTid;
   if(nBlk * nTid < size){
      nBlk +=1;
   }
   int * arr_d;
   int* histogram_d;
   hipMalloc((void**)&arr_d,size*sizeof(int));
   hipMemcpy(arr_d,arr,size*sizeof(int),hipMemcpyHostToDevice);
   hipMalloc((void**)&histogram_d,max_val*sizeof(int));
   hipMemset((void**)&histogram_d,0,max_val*sizeof(int));
   //cuda_sort<<<nBlk,nTid>>>(arr_d,histogram_d, size, max_val);
   //cuda_sort<<<1,size,size*sizeof(int)>>>(arr_d,histogram_d, size, max_val);
   cuda_sort<<<1,size>>>(arr_d,histogram_d, size, max_val);
   // copy to host & finsh
   hipMemcpy(arr,arr_d,size*sizeof(int),hipMemcpyDeviceToHost);
   hipFree(arr_d); hipFree(histogram_d);
}
__global__ void cuda_sort(int* arr_d,int* histogram_d, int size, int max_val){
   //extern __shared__ int position[]; // https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
   int i = threadIdx.x + blockDim.x * blockIdx.x;
   //__device__ int histogram_d[max_val];
   int off = 1;
   if(i<size){
      atomicAdd(&histogram_d[arr_d[i]],1);
   }
   __syncthreads();
   // https://www.eecs.umich.edu/courses/eecs570/hw/parprefix.pdf
   if(i<size){
      position[i] = histogram_d[i];
   }
   for(int stride = size>>1; stride>0; stride >>=1){
      __syncthreads();
      if(i<stride){
         position[off*(2*i+2)-1] += position[off*(2*i+1)-1];
      }
      off *=2;
   }
   __syncthreads();
   for(int j=off-1; j<size-1; j++){
      if(i == j){
         position[j+1] += position[j];
      } 
      __syncthreads();
   }
   if(i==0) position[off-1] = 0;
   for(int stride= 1; stride<size; stride *=2){
      off = off>>1;
      __syncthreads();
      if(i<stride){
         int a = off*(2*i+1)-1;
         int b = off*(2*i+2)-1;
         int tmp = position[a];
         position[a] = position[b];
         position[b] += tmp;
      }
   }
   __syncthreads();
   
   if(i<size){
      //arr_d[i] = histogram_d[i];
      //arr_d[i] = position[i];
      for(int j=0; j<max_val; j++){
         for(int k=0; k<histogram_d[j]; k++){
            arr_d[position[j]+k] = j;
         }
      }
   }
   // device code
}
/*
   atomicAdd(&value[I],1);
   int index =0;
   int i,j;
   for(i=0; i<max_val; i++){
      histogram_d[i] =0;
   }
   for (i=0; i<size; i++){
      histogram_d[arr_d[i]]++;
   }

   for(i=0; i<max_val; i++){
      for(j=0; j<histogram_d[i]; j++){
         arr_d[index++] = i;
      }
   }
*/

