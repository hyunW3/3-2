#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int n, float* x, float* y){
    for(int i = 0; i < n; i++){
        y[i] = x[i] + y[i];
    }

}

int main(){

    int N = 1000;
    int i, j, k;
    float* x, *x_d;
    float* y, *y_d;
    int size = N * sizeof(float);
    float maxError = 0.0;

    x = (float*)malloc(sizeof(float)*N);
    y = (float*)malloc(sizeof(float)*N);
    hipMalloc(&x_d, N*sizeof(float));
    hipMalloc(&y_d, N*sizeof(float));

    for(i = 0; i < N; i++){
        x[i] = 1.0;
        y[i] = 2.0;
    }
    hipMemcpy(x_d, x, size, hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, size, hipMemcpyHostToDevice);

    add<<<10, 10>>>(N, x_d, y_d);

    hipDeviceSynchronize();
    hipMemcpy(y, y_d, size, hipMemcpyDeviceToHost);

    for(i = 0; i < N; i++) {
	printf("y[%d]: %f\n", i, y[i]);
        maxError = max(maxError, abs(y[i]-3.0));
    }

    printf("max Error %9.5lf\n", maxError);

    hipFree(x_d);
    hipFree(y_d);
    return 0;

}
